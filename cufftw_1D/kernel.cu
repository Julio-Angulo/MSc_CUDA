//Calculo de la FFT 1D usando "fftwf_plan_dft_1d".

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>


#define SIGNAL_SIZE 10



int main()
{
	int i;
	hipFloatComplex *h_xn;
	hipFloatComplex *h_Xk;
	fftwf_complex *in,*out;

	//Se reserva memoria para h_xn en el host
	h_xn = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*SIGNAL_SIZE);

	//Se reserva memoria para h_Xk en el host
	h_Xk = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*SIGNAL_SIZE);

	//Se dan valores a x[n]
	for(i=0;i<SIGNAL_SIZE;i++)
	{
		
			//h_xn[i] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%21));
			h_xn[i] = make_hipFloatComplex((float)(i+1),(float)(0.0));
	}

	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(i = 0; i<SIGNAL_SIZE;i++)
	{
		printf(" x[%d] = (%f) + (%f)\n",i,hipCrealf(h_xn[i]),hipCimagf(h_xn[i]));
	}

	//Se reserva memoria para "in" en el device
	hipMalloc((void**)&in,sizeof(hipfftComplex)*SIGNAL_SIZE);

	//Se reserva memoria para "out" en el device
	hipMalloc((void**)&out,sizeof(hipfftComplex)*SIGNAL_SIZE);

	//Se copian los datos de h_xn >>> in 
	hipMemcpy(in,h_xn,sizeof(hipFloatComplex)*SIGNAL_SIZE,hipMemcpyHostToDevice);

	//CUFFT plan
	fftw_plan plan;
	//cufftHandle plan;
	plan = fftwf_plan_dft_1d(SIGNAL_SIZE,in,out,FFTW_FORWARD,FFTW_ESTIMATE);
	//cufftPlan1d(&plan, SIGNAL_SIZE, CUFFT_C2C, 1);

	//Ejecucion de la fft
	fftwf_execute(plan);

	//Se copian los datos de out >>> h_Xk
	hipMemcpy(h_Xk,out,sizeof(hipfftComplex)*SIGNAL_SIZE,hipMemcpyDeviceToHost);

	//Se imprimen los valores de salida X[k]
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(i = 0; i<SIGNAL_SIZE;i++)
	{
		printf(" X[%d] = (%f) + (%f)\n",i,hipCrealf(h_Xk[i]),hipCimagf(h_Xk[i]));
	}

	//Se destruye el plan
	fftwf_destroy_plan(plan);

	//Se liberan memorias
	free(h_xn);
	free(h_Xk);
	hipFree(in);
	hipFree(out);
}
