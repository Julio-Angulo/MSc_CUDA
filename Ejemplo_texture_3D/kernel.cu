#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>

//declare texture reference
texture<float,hipTextureType3D,hipReadModeElementType> textReference_A;
texture<float,hipTextureType3D,hipReadModeElementType> textReference_B;

// kernel function
__global__ void kernel(float *C_device, int alto, int ancho, int prof)
{
	int xIndex;
	int yIndex;
	int zIndex;

	//calculate each thread global index
	xIndex = threadIdx.x + blockIdx.x * blockDim.x;
	yIndex = threadIdx.y + blockIdx.y * blockDim.y;
	zIndex = threadIdx.z + blockIdx.z * blockDim.z;

	
	C_device[(zIndex*alto*ancho) + (yIndex*ancho) + xIndex] = tex3D(textReference_A,xIndex,yIndex,zIndex) + tex3D(textReference_B,xIndex,yIndex,zIndex);
		
}

int main(int argc, char* argv[])
{
	float *A_host;
	float *B_host;
	float *C_host;
	float *C_device;
	hipArray *cudaArray_A;
	hipArray *cudaArray_B;
	hipExtent volumeSize_A;
	hipExtent volumeSize_B;
	hipChannelFormatDesc channel_A;
	hipChannelFormatDesc channel_B;
	hipMemcpy3DParms copyparms_A={0};
	hipMemcpy3DParms copyparms_B={0};
	
	int alto,ancho,prof;

	alto = atoi(argv[1]);
	ancho = atoi(argv[2]);
	prof = atoi(argv[3]);

	//allocate host and device memory
	A_host = (float*)malloc(sizeof(float)*alto*ancho*prof);
	B_host = (float*)malloc(sizeof(float)*alto*ancho*prof);
	C_host = (float*)malloc(sizeof(float)*alto*ancho*prof);
	hipMalloc((void**)&C_device,sizeof(float)*alto*ancho*prof);

	//initialize A_host matrix before usage
	for(int loop=0; loop<alto*ancho*prof;loop++)
		A_host[loop] = (float)(rand() % 3);
	
	//initialize B_host matrix before usage
	for(int loop=0; loop<alto*ancho*prof;loop++)
		B_host[loop] = (float)(rand() % 5);
    

	printf("A_host:\n");
	for(int i = 0;i<prof;i++)
	{
		for(int j = 0;j<alto;j++)
		{
			for(int k = 0;k<ancho;k++)
			{
				printf(" %f ",A_host[i*alto*ancho + j*ancho + k]);
			}
			printf("\n");
		}
		printf("\n\n");
	}
	
	printf("\n");
	printf("B_host:\n");
	for(int i = 0;i<prof;i++)
	{
		for(int j = 0;j<alto;j++)
		{
			for(int k = 0;k<ancho;k++)
			{
				printf(" %f ",B_host[i*alto*ancho + j*ancho + k]);
			}
			printf("\n");
		}
		printf("\n\n");
	}
	

	//set cuda array volume size
	volumeSize_A = make_hipExtent(ancho,alto,prof);
	volumeSize_B = make_hipExtent(ancho,alto,prof);

	//create channel to describe data type
	channel_A = hipCreateChannelDesc<float>();
	channel_B = hipCreateChannelDesc<float>();

	//allocate device memory for cuda array
	hipMalloc3DArray(&cudaArray_A,&channel_A,volumeSize_A);
	hipMalloc3DArray(&cudaArray_B,&channel_B,volumeSize_B);

	//set cuda array copy parameters
	copyparms_A.extent = volumeSize_A;
	copyparms_A.dstArray = cudaArray_A;
	copyparms_A.kind = hipMemcpyHostToDevice;
	copyparms_B.extent = volumeSize_B;
	copyparms_B.dstArray = cudaArray_B;
	copyparms_B.kind = hipMemcpyHostToDevice;

	// 3D copy from host_CubeMatrix to hipArray
	copyparms_A.srcPtr = make_hipPitchedPtr((void*)A_host,ancho*sizeof(float),ancho,alto);
	hipMemcpy3D(&copyparms_A);
	copyparms_B.srcPtr = make_hipPitchedPtr((void*)B_host,ancho*sizeof(float),ancho,alto);
	hipMemcpy3D(&copyparms_B);

	//set texture filter mode property
	//use hipFilterModePoint of hipFilterModeLinear
	textReference_A.filterMode = hipFilterModePoint;
	textReference_B.filterMode = hipFilterModePoint;

	//set texture address mode property
	//use hipAddressModeClamp or hipAddressModeWrap for integer coordinates
	textReference_A.addressMode[0] = hipAddressModeClamp;
	textReference_A.addressMode[1] = hipAddressModeClamp;
	textReference_A.addressMode[2] = hipAddressModeClamp;
	textReference_B.addressMode[0] = hipAddressModeClamp;
	textReference_B.addressMode[1] = hipAddressModeClamp;
	textReference_B.addressMode[2] = hipAddressModeClamp;

	//bind texture reference with cuda array
	hipBindTextureToArray(textReference_A,cudaArray_A, channel_A);
	hipBindTextureToArray(textReference_B,cudaArray_B, channel_B);

	// preparing kernel launch
	dim3 blockDim; dim3 gridDim;
	blockDim.x = ancho; blockDim.y = alto; blockDim.z = prof;
	gridDim.x = 1; gridDim.y = 1; gridDim.z = 1;

	//execute device kernel
	kernel<<< gridDim , blockDim >>>(C_device, alto, ancho, prof);
	hipDeviceSynchronize();

	//unbind texture reference to free resource
	hipUnbindTexture(textReference_A);
	hipUnbindTexture(textReference_B);

	//copy result matrix from device to host memory
	hipMemcpy(C_host, C_device, sizeof(float)*alto*ancho*prof, hipMemcpyDeviceToHost);
	
	

	printf("\n");
	printf("C_host:\n");
	for(int i = 0;i<prof;i++)
	{
		for(int j = 0;j<alto;j++)
		{
			for(int k = 0;k<ancho;k++)
			{
				printf(" %f ",C_host[i*alto*ancho + j*ancho + k]);
			}
			printf("\n");
		}
		printf("\n\n");
	}
	
	//free host and device memory
	free(A_host);
	free(B_host);
	free(C_host);
	hipFree(C_device);
	hipFreeArray(cudaArray_A);
	hipFreeArray(cudaArray_B);
}