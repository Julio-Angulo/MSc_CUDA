///�sta programa calcula la versi�n paralelizada del algoritmo FFT_DIF_DIT_TD
///(04/01/2017)
///�sta versi�n sirve para graficar en matlab los errores absolutos y relativos Caso: N^13, Li=264, Lo=264

#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <hip/hip_math_constants.h>
#include <iostream>
#include <time.h>

//////////////////////////////////////////////////////////////////////////
///////////////////////DECLARACI�N DE FUNCIONES///////////////////////////
//////////////////////////////////////////////////////////////////////////
void vector_entrada_xn(int Li);
void arreglo_W(int N);
void asign_rap(int N,int Li,int Lo);
void factor(int N);
void product(int vector_1[50],int vector_2[50],int valor);
void etapa_entrada(void);
__global__ void inputStage_kernel(int N, int Li,int Dip,int Dop,int P,hipFloatComplex *x,hipFloatComplex *W,hipFloatComplex *y);
void etapa_intermedia(void);
void etapa_salida(void);
__global__ void outputStage_kernel(int N,int Lo,int Dip,int Dop,int P,hipFloatComplex *z,hipFloatComplex *W,hipFloatComplex *X);

//////////////////////////////////////////////////////////////////////////
/////////////////////DECLARACI�N DE VARIABLES GLOBALES////////////////////
//////////////////////////////////////////////////////////////////////////
hipFloatComplex *x_host;
hipFloatComplex *W_host;
//cuFloatComplex *y_host;
//cuFloatComplex *z_host;
hipFloatComplex *X_host;
hipFloatComplex *x_device;
hipFloatComplex *W_device;
hipFloatComplex *y_device;
hipFloatComplex *z_device;
hipFloatComplex *X_device;
hipfftComplex *in,*out;
FILE *db_open,*dc_open;

int Dip,Dop,P,N,Li,Lo;
int vF[50]; //Almacena los factores de N
int svF; //Almacena el numero de factores de N
int Prod[50];
int a;

#define inf 99999

//////////////////////////////////////////////////////////////////////////
//////////////////////////DATOS DE ENTRADA////////////////////////////////
//////////////////////////////////////////////////////////////////////////
/// N  >>> N�mero de elementos del vector de entrada
/// Li >>> N�mero de elementos de entrada diferentes de cero
/// Lo >>> N�mero de elementos de salida requeridos
/// loop >>> N�mero de iteraciones
/// muestras >>> N�mero de muestras

//////////////////////////////////////////////////////////////////////////
///////////////////////////DATOS DE SALIDA////////////////////////////////
//////////////////////////////////////////////////////////////////////////
/// X >>> Vector de salida

//////////////////////////////////////////////////////////////////////////
/////////////////// SE INGRESAN LOS DATOS DE ENTRADA /////////////////////
//////////////////////////////////////////////////////////////////////////

///Ingrese el n�mero de iteraciones requeridas
const int loop = 300;

///Ingrese el valor de N_max

const int N_max = 13;

///Ingrese el valor de Li_max
const int Li_max = 264;

///Ingrese el valor de Lo_max
const int Lo_max = 264;

//////////////////////////////////////////////////////////////////////////
//////////////////////////FUNCION PRINCIPAL///////////////////////////////
//////////////////////////////////////////////////////////////////////////

//Funci�n principal 
int main()
{
	int i,j,i_N,j_res,k_res,cont,i_prom,m;
	float *parte_real;
    float *parte_imag;
	//float suma;
	//float promedio[N_max];

	FILE *da,*db;
	//da = fopen("Tiempos_N20_LiN_LoVARIA_CUDA.bin","a+b"); //Crea o sobre escribe archivo
	da = fopen("Resultados_N13_Li264_Lo264_real_CUDA.bin","a+b"); //Crea o sobre escribe archivo
    db = fopen("Resultados_N13_Li264_Lo264_imag_CUDA.bin","a+b"); //Crea o sobre escribe archivo


    //Pausa
	printf("\n---PRESIONA UNA TECLA PARA CONTINUAR---\n\n");
	getchar();
	
	for(i_N = N_max;i_N <= N_max;i_N++)
    {
        N = (int )pow(3,i_N);
        printf("\n N = %d \n",N);

		for(j_res=Li_max;j_res <= Li_max;j_res++)
        {
            Li=j_res;
			for(k_res=Lo_max;k_res <= Lo_max;k_res++)
            {
				Lo=k_res;
				printf("\n Li = %d  Lo = %d",Li,Lo);

				//////////////////////////////////////////////////////////
				parte_real = (float*) malloc(Lo*sizeof(float));
				parte_imag = (float*) malloc(Lo*sizeof(float));

				for(m=0;m<=Lo-1;m++)
                {
					parte_real[m] = 0.0;
                    parte_imag[m] = 0.0;

                }

				///Se abre el archivo binario
                db_open = fopen("Entrada_real_N13_C.bin","rb");
                dc_open = fopen("Entrada_imag_N13_C.bin","rb");

				//suma=0.0;
				for(j=0;j<loop;j++)
				{
		
					//Comandos necesarios para medir el tiempo
					float elapsedTime_app;
					hipEvent_t start_app, stop_app;
					hipEventCreate(&start_app);
					hipEventCreate(&stop_app);

					//Se generan en el host los valores del vector de entrada x[n] 
					vector_entrada_xn(Li);

					///Se genera el arreglo W[N]
					arreglo_W(N);

					//---------------------------------------------------------------------------------------------
					//Se empieza a medir el tiempo de ejecucion de la aplicacion
					hipEventRecord(start_app,0);

					//Se generan en el host los factores Dip y Dop
					asign_rap(N,Li,Lo);

					//C�lculo en el host del factor P
					P = N/(Dip*Dop);

					//printf("\n\n FACTOR P:\n\n");
					//printf("\n Dip = %d Dop = %d P = %d ",Dip,Dop,P);

					//Funci�n auxiliar del host para ejecutar la etapa de entrada
					etapa_entrada();

					//Funci�n auxiliar del host para ejecutar la etapa intermedia
					etapa_intermedia();

					//Funci�n auxiliar del host para ejecutar la etapa de salida
					etapa_salida();

					////////////////////////////////////////////////////////////////////////////////////////////
					////////////////////////////////////////////////////////////////////////////////////////////
                    ///SUMATORIAS
					for(m=0;m<=Lo-1;m++)
                    {
						parte_real[m] = parte_real[m] + hipCrealf(X_host[m]);
                        parte_imag[m] = parte_imag[m] + hipCimagf(X_host[m]);
                        //printf("\n X[%d] = %.4f + (%.4f)",m,creal(X[m]),cimag(X[m]));
                        //fprintf(dc,"%f %f\n",creal(X[m]),cimag(X[m]));
                    }


					////////////////////////////////////////////////////////////////////////////////////////////
					////////////////////////////////////////////////////////////////////////////////////////////

					//---------------------------------------------------------------------------------------------
					//Comandos necesarios para medir el tiempo de la aplicacion (app)
					hipEventRecord(stop_app,0);
					hipEventSynchronize(stop_app);
					hipEventElapsedTime(&elapsedTime_app,start_app,stop_app);

					//Suma de todos los tiempos
					//suma = suma + elapsedTime_app;

					//Se destruyen los eventos que miden el tiempo de la aplicacion
					hipEventDestroy(start_app);
					hipEventDestroy(stop_app);

					//Se liberan memorias del Host y Device
					free(x_host);
					free(W_host);
					free(X_host);
					hipFree(x_device);
					hipFree(W_device);
					hipFree(y_device);
					hipFree(z_device);
					hipFree(X_device);

				}
				///////////////////////////////////
				///PROMEDIO DE ERRORES
				for(m=0;m<=Lo-1;m++)
                {
					parte_real[m] = parte_real[m]/loop;
					parte_imag[m] = parte_imag[m] /loop;
                }
				//////////////////////////////////
				///Se imprimen los resultados en los archivos binarios
				fwrite(parte_real,sizeof(float),Lo,da);
                fwrite(parte_imag,sizeof(float),Lo,db);
				//promedio[k_res-1] = suma/(float)loop;
				fclose(db_open);
                fclose(dc_open);
				free(parte_real);
				free(parte_imag);
			}
			
		}
	}
	//fwrite(promedio,sizeof(float),N_max,da);
    fclose(da);	
	fclose(db);	

	return EXIT_SUCCESS;
}

//////////////////////////////////////////////////////////////////////////
/////////////////////////FUNCIONES SECUNDARIAS////////////////////////////
//////////////////////////////////////////////////////////////////////////

//�sta funci�n genera el vector de entrada x[n]
void vector_entrada_xn(int Li)
{
	//Declaraci�n de variables locales
	int k;
	float *buffer_real,*buffer_imag;

	//Se reserva memoria para xn_host en el host
	x_host = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*N);
	buffer_real = (float*)malloc(sizeof(float)*(pow(3,N_max)));
	buffer_imag = (float*)malloc(sizeof(float)*(pow(3,N_max)));

	///Se lee el vector de entrada del archivo binario
	fread(buffer_real,sizeof(float),(int)pow(3,N_max),db_open);
    fread(buffer_imag,sizeof(float),(int)pow(3,N_max),dc_open);


	//Se dan valores a x[n]
	for(k = 0;k < N; k++)
	{
		//x_host[k] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%11));
		//x_host[k] = make_cuFloatComplex((float)(k + 1),(float)(0.0));
		x_host[k] = make_hipFloatComplex(buffer_real[k],buffer_imag[k]);
	}
	


	/*
	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(k=0;k<Li;k++) 
	{
		printf(" %d-> (%f) + (%f)\n",k+1,cuCrealf(x_host[k]),cuCimagf(x_host[k]));
	}
	*/
	
	free(buffer_real);
	free(buffer_imag);
}


//�sta funci�n genera el arreglo W
void arreglo_W(int N)
{
	//Declaraci�n de variables locales
	int n;

	//Se reserva memoria para W_host en el host
	W_host = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*N);

	//Se genera el arreglo W
	for(n = 1;n <= N;n++)
	{
		W_host[n-1] = make_hipFloatComplex((float)cos((2*HIP_PI*n)/N),(float)(-1)*sin((2*HIP_PI*n)/N));
	}
	
	/*
	//Se imprimen los valores del arreglo W[N]
	printf("\n---ARREGLO W[N]---\n\n");
	for(n = 0;n < N; n++)
	{
		printf(" W[%d]-> (%f) + (%f)\n",n+1,cuCrealf(W_host[n]),cuCimagf(W_host[n]));
	}
	*/

}

//�sta funci�n genera los factores Dip y Dop
void asign_rap(int N,int Li,int Lo)
{
	//Declaraci�n de variables locales
	float NLi,NLo,Diprapt,Doprapt;
	int Nh[50];
	int k[50];
	int G;
	int g,i,t,ta;
	int Dipt[50],Dopt[50];
	float distrapt,distrap;
	int Pos,h,Poss;
	int nk[50];
	int r;

	//Inicializaciones
	G = 0;
	svF = 0;


	//Factores Dip y Dop ideales
	NLi=(float)N/(float)Li;
    NLo=(float)N/(float)Lo;
    Diprapt=NLi;
    Doprapt=NLo;

	//Se encuentran los factores de "N"
	//vF almacena los factores de "N"
	//svF almacena el n�mero de factores de "N"
	factor(N);

	/*
	Almacena en el vector Nh los factores que son diferentes de del vector vF
    En el vector k se almacena la cantidad de veces que se repite cada
    elemento almacenado en el vector Nh.
	*/

    Nh[0] = vF[0];
    k[0]=1;
	for(g=1;g<=svF-1;g=g+1)
    {
        if(vF[g]!=vF[g-1])
        {
           G=G+1;
           Nh[G]=vF[g];
           k[G]=1;
        }
        else
        {
            k[G]=k[G]+1;
        }

    }

	/*
	Almacena en el vector Nh todas las posibles combinaciones que den como
    producto a N. t almacena el numero de elementos del vector Nh.
	*/
	product(Nh,k,G);
	t = a;
	for(i=0;i<t;i=i+1)
    {
        Dipt[i]=Prod[i];
    }

	distrapt=inf;

	for(g=1;g<=t;g=g+1)
    {

        if(Dipt[g-1]<=NLi)
        {

            Pos=g-1;
            for(h=0;h<=G;h=h+1)
            {
                Poss=floor(Pos/(k[h]+1));
                nk[h]=k[h]+Poss*(k[h]+1)-Pos;
                Pos=Poss;
            }

            product(Nh,nk,G);

            ta=a;
            for(i=0;i<ta;i=i+1)
                {
                   Dopt[i]=Prod[i];
                }
          ////////////////////////////////////////////
           //int j;
           //for(j=0;j<ta;j++)
           //{
           //    printf(" %d ",Dopt[j]);
           //}
           //printf("\n\n ta=%d\n\n",ta);
           ///////////////////////////////////////////
            for(r=0;r<ta;r=r+1)
                {
                    distrap=sqrt(pow(Diprapt-(Dipt[g-1]),2)+pow(Doprapt-(Dopt[r]),2));
                    if(distrap<distrapt)
                        {
                            distrapt=distrap;
                            Dip=Dipt[g-1];
                            Dop=Dopt[r];
                        }
                }

        }

    }

	/*
	printf("\n\n FACTOR Dip :\n\n");
	printf(" %d ",Dip);

	printf("\n\n FACTOR Dop:\n\n");
	printf(" %d ",Dop);
	*/

}

//�sta funci�n encuentra los factores de "N"
void factor(int N)
{
	//Se empieza a verificar los factores desde 2
	int i=2;
	long N_factor;
	N_factor = N;
	 while(i<=N_factor)
	{
      while((N_factor%i)==0)
      {
          vF[svF]=i;
          N_factor=N_factor/i;
         // printf("Factores: %d ",vF[svF]);
          svF++;
      }
	i++;
	}

}

//�sta funci�n encuentra todas las posibles combinaciones de factores que den como resultado "N"
void product(int vector_1[50],int vector_2[50],int valor)
{
	int d,e,s,pNh,i;
    int cont=0;
    Prod[0]=1;
    a=1;
    for(d=0;d<=valor;d=d+1)
    {

        s=a;
        pNh=1;
        for(e=1;e<=vector_2[d];e=e+1)
        {
            pNh=pNh*vector_1[d];

            for(i=(s*e+1);i<=(s*e+s);i=i+1)
            {
                Prod[i-1]=pNh*Prod[cont];
                cont=cont+1;
            }
            a=a+s;
            cont=0;
        }

    }


}

//Funci�n auxiliar del host para calcular la etapa de entrada en el device
void etapa_entrada(void)
{
	//////////////////////////////////////////////////////////////////////////
	////////////////////////////ETAPA DE ENTRADA//////////////////////////////
	//////////////////////////////////////////////////////////////////////////

	//Declaraci�n de variables locales
	int k1,n1,n2;

	//Asignaci�n de memoria en el device para el arreglo "x_device"
	hipMalloc((void**)&x_device,Li*sizeof(hipFloatComplex));

	//Se reserva memoria en el device para el arreglo "W_device"
	hipMalloc((void**)&W_device,N*sizeof(hipFloatComplex));

	//Asignaci�n de memoria en el device para el arreglo "y"
	hipMalloc((void**)&y_device,P*Dip*Dop*sizeof(hipFloatComplex));

	//Se pasa el arreglo x_host a x_device
	hipMemcpy(x_device,x_host,Li*sizeof(hipFloatComplex),hipMemcpyHostToDevice);

	//Env�o de los arreglos W hacia la memoria global del device
	hipMemcpy(W_device,W_host,N*sizeof(hipFloatComplex),hipMemcpyHostToDevice);

	//Asignaci�n de memoria en el host para "y"
	//y_host = (cuFloatComplex*)malloc(sizeof(cuFloatComplex)*P*Dip*Dop);

	//Dimensionamiento del grid para la funci�n kernel "inputStage"
	//Dimensionamiento del Grid
	dim3 gridDim(1,1,1);
	//Dimensionamiento del block
	dim3 blockDim(1,1,1);
	if((P*Dop) < 32 && (Dip) < 32)
	{
		blockDim.x = (P*Dop);
		blockDim.y = (Dip);
		gridDim.x = 1;
		gridDim.y = 1;
	}
	else
	{
		blockDim.x = 32;
		blockDim.y = 32;
		gridDim.x = (unsigned int) (ceilf((float)(P*Dop)/(float)blockDim.x));
		gridDim.y = (unsigned int) (ceilf((float)Dip/(float)blockDim.y));
	}

	//Lanzamiento del kernel "inputStage_kernel"
	inputStage_kernel<<<gridDim,blockDim>>>(N,Li,Dip,Dop,P,x_device,W_device,y_device);

	//Esperar que el kernel termine de ejecutarse totalmente
	hipDeviceSynchronize();

	/*
	//Copia del arreglo "y" del device hacia el host
	cudaMemcpy(y_host,y_device,sizeof(cuFloatComplex)*P*Dip*Dop,cudaMemcpyDeviceToHost);
	
	
	//Se imprimen los valores de "y"
	printf("\n\n--- ARREGLO y(n1,n2,k1) ---\n\n");
	for(k1 = 0;k1 < Dip;k1++) 
	{
		for(n1 = 0;n1 < Dop;n1++)
		{
			for(n2 = 0;n2 < P;n2++)
			{
				printf(" (%f) + (%f) ",cuCrealf(y_host[(k1*Dop*P)+(n1*P)+n2]),cuCimagf(y_host[(k1*Dop*P)+(n1*P)+n2]));
			}
			printf("\n");
		}
		printf("\n\n");
	}
	printf("\n");
	*/
	
}

//funci�n kernel que ejecuta la etapa de entrada en el device
__global__ void inputStage_kernel(int N, int Li,int Dip,int Dop,int P,hipFloatComplex *x,hipFloatComplex *W,hipFloatComplex *y)
{
	int n1,n2;
	hipFloatComplex t1;

	//Threads
	int n = blockDim.x *blockIdx.x + threadIdx.x;
	int k1 = blockDim.y *blockIdx.y + threadIdx.y;

	//Se resetean las flags
	//flag_inputstage_1_d[0] = 0;
	//flag_inputstage_2_d[0] = 0;
	//flag_inputstage_3_d[0] = 0;

	//printf("\n n = %d k1 = %d",n,k1);

	if( (n < (P*Dop)) && (k1 < Dip))
	{
		n2 = floorf(n/Dop);
		n1 = n - (Dop*n2);
		//Generaci�n de los elementos que dependen de x[0]
		if(n == 0)
		{
			y[(k1*Dop*P)+(0*P)+ 0] = x[0];

			///Flag
			//flag_inputstage_1_d[0] = 1;
			
		}
		//Mapeo de x[n] a las entradas del primer conjunto de Dop DFT's
		if((n >= 1) && (n <= (Li-1)))
		{
			t1 = x[n];
			if(k1 == 0)
			{
				y[(0*Dop*P)+(n1*P)+ n2] = t1;
			}
			if(k1 >= 1)
			{
				y[(k1*Dop*P)+(n1*P)+ n2] = hipCmulf(W[((n*k1)%N)-1],t1);
			}

			///Flag
			//flag_inputstage_2_d[0] = 1;
		}
		//Rellenado de ceros para los elementos de "y" para Li <= n <= (P*Dop)-1
		if((n >= Li) && (n <= (P*Dop)-1))
		{
			y[(k1*Dop*P)+(n1*P)+ n2] = make_hipFloatComplex(0.0,0.0);

			///Flag
			//flag_inputstage_3_d[0] = 1;
		}

		
		//printf("\n (%f) + (%f)\n ",cuCrealf(y[(k1*Dop*P)+(n1*P)+ n2]),cuCimagf(y[(k1*Dop*P)+(n1*P)+ n2]));
	}
}

//Funci�n auxiliar del host para calcular la etapa intermedia en el device
void etapa_intermedia(void)
{
	//////////////////////////////////////////////////////////////////////////
	////////////////////////////ETAPA INTERMEDIA//////////////////////////////
	//////////////////////////////////////////////////////////////////////////

	//Declaraci�n de variables locales
	int k1,k2,n1;
	int n[1] = {P};
	int inembed[1] = {P};
	int onembed[1] = {P};

	//Asignaci�n de memoria en el device para "z"
	hipMalloc((void**)&z_device,P*Dip*Dop*sizeof(hipFloatComplex));

	//Asignaci�n de memoria en el host para "z"
	//z_host = (cuFloatComplex*)malloc(sizeof(cuFloatComplex)*P*Dip*Dop);

	//Asignaci�n de memoria en el device para "in" y "out"
	hipMalloc((void**)&in,sizeof(hipfftComplex)*P*Dip*Dop);
	hipMalloc((void**)&out,sizeof(hipfftComplex)*P*Dip*Dop);

	//Se copia el arreglo "y" al arreglo "in"
	hipMemcpy(in,y_device,sizeof(hipFloatComplex)*P*Dip*Dop,hipMemcpyDeviceToDevice);

	//Se crea un plan
	hipfftHandle plan;
	hipfftPlanMany(&plan,1,n,inembed,1,P,onembed,1,P,HIPFFT_C2C,Dip*Dop);

	//Ejecuci�n del plan
	hipfftExecC2C(plan,in,out,HIPFFT_FORWARD);

	//Esperar que el kernel termine de ejecutarse totalmente
	hipDeviceSynchronize();

	//Se copian los datos del arreglo "out" al arreglo "z_device"
	hipMemcpy(z_device,out,sizeof(hipfftComplex)*P*Dip*Dop,hipMemcpyDeviceToDevice);

	//Se destruye el plan
	hipfftDestroy(plan);

	//Se liberan los arreglos "in" y "out"
	hipFree(in);
	hipFree(out);

	/*
	//Se copian los datos del arreglo "z_device" al arreglo "z_host"
	cudaMemcpy(z_host,z_device,sizeof(cuFloatComplex)*P*Dip*Dop,cudaMemcpyDeviceToHost);

	
	///Se imprimen los valores de z(n1,k2,k1)
	printf("\n\n--- ARREGLO z(n1,k2,k1) ---\n\n");
	for(k1 = 0;k1 < Dip;k1++) 
	{
		for(n1 = 0;n1 < Dop;n1++)
		{
			for(k2 = 0;k2 < P;k2++)
			{
				printf(" (%f) + (%f) ",cuCrealf(z_host[(k1*Dop*P)+(n1*P)+k2]),cuCimagf(z_host[(k1*Dop*P)+(n1*P)+k2]));
			}
			printf("\n");
		}
		printf("\n\n");
	}
	printf("\n");
	*/
	
}

//Funci�n auxiliar del host para calcular la etapa de salida en el device
void etapa_salida(void)
{
	//////////////////////////////////////////////////////////////////////////
	////////////////////////////ETAPA DE SALIDA///////////////////////////////
	//////////////////////////////////////////////////////////////////////////

	//Declaraci�n de variables locales
	int m;

	//Asignaci�n de memoria en el device para "X"
	hipMalloc((void**)&X_device,Lo*sizeof(hipFloatComplex));

	//Asignaci�n de memoria en el host para "X"
	X_host = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*Lo);

	//Dimensionamiento del grid para la funci�n kernel "outputStage"
	//Dimensionamiento del Grid
	dim3 gridDim(1,1,1);
	//Dimensionamiento del block
	dim3 blockDim(1,1,1);
	if((Lo) < 1024)
	{
		blockDim.x = Lo;
		gridDim.x = 1;
	}
	else
	{
		blockDim.x = 1024;
		gridDim.x = (unsigned int) (ceilf((float)Lo/(float)blockDim.x));
	}

	//Lanzamiento del kernel "outputStage_kernel"
	outputStage_kernel<<<gridDim,blockDim>>>(N,Lo,Dip,Dop,P,z_device,W_device,X_device);

	//Esperar que el kernel termine de ejecutarse totalmente
	hipDeviceSynchronize();

	//Copia del arreglo "X" del device hacia el host
	hipMemcpy(X_host,X_device,sizeof(hipFloatComplex)*Lo,hipMemcpyDeviceToHost);
			
	/*
	//Se imprimen los valores de "X_host"
    ///Imprimir X[k]
    printf("\n\n--- ARREGLO X[k] ---\n\n");
    for(m=0;m<=Lo-1;m++)
    {
        printf("\n X[%d] = %.4f + (%.4f)",m,cuCrealf(X_host[m]),cuCimagf(X_host[m]));
        //fprintf(da,"%.4f %.4f\n",creal(X[i]),cimag(X[i]));
    }
	*/
	

}

//funci�n kernel que ejecuta la etapa de salida en el device
__global__ void outputStage_kernel(int N,int Lo,int Dip,int Dop,int P,hipFloatComplex *z,hipFloatComplex *W,hipFloatComplex *X)
{
	//Declaraci�n de variables locales
	int n1,k_aux,k1,k2,a,b;
	hipFloatComplex t1,t2,t3,t4,t5;
	

	//Threads
	int k = blockDim.x *blockIdx.x + threadIdx.x;

	//Se resetean las flags
	//flag_outputstage_1_d[0] = 0;
	//flag_outputstage_2_d[0] = 0;
	//flag_outputstage_3_d[0] = 0;

	if(k < Lo)
	{
		for(n1 = 0; n1 <= (Dop-1); n1 = n1+1)
		{
			if(Lo <= Dip)
			{
				//C�lculo de X(k) para 0<=k<=Lo-1.
				//printf("\n--- Caso (Lo <= Dip) ---\n");
				//En la descomposici�n k = k1 + Dipk2; k2 = 0, y por lo tanto, k = k1
				if(n1 == 0) //Caso para lograr que por lo menos ingrese una vez 
				{
					X[k] = z[(k*Dop*P)+(0*P) + 0];
					
					///Flag
					//flag_outputstage_1_d[0] = 1;
				}
				else
				{
					if(n1 == 1)
					{
						X[k] = z[(k*Dop*P)+(0*P) + 0];
					}
					X[k] = hipCaddf(z[(k*Dop*P)+(n1*P) + 0],X[k]);

					///Flag
					//flag_outputstage_1_d[0] = 1;

				}

			}
			else
			{
				if((k >= 0) && (k <= (Dip-1)))
				{
					//C�lculo de X(k) para 0<=k<=Dip-1.
					//En la descomposici�n k = k1 + Dipk2; k2 = 0, y por lo tanto, k = k1
					if(n1 == 0) //Caso para lograr que por lo menos ingrese una vez 
					{
						X[k] = z[(k*Dop*P)+(0*P) + 0];
					}
					else
					{
						if(n1 == 1)
						{
							X[k] = z[(k*Dop*P)+(0*P) + 0];
						}
						X[k] = hipCaddf(z[(k*Dop*P)+(n1*P) + 0],X[k]);

					}
					

				}
				else
				{
					
					if(Dop <= 4)
					{
						//Usando el m�todo directo
						//printf("\n--- Caso (Metodo directo) ---\n");
						if(n1 == 0) //Caso para lograr que por lo menos ingrese una vez 
						{
							k_aux = k-((Dip*P)*floorf(k/(Dip*P)));
							k2 = floorf(k_aux/Dip);
							k1 = k_aux-(Dip*k2);
							X[k] = z[(k1*Dop*P)+(0*P)+ (k2%P)];

							///Flag
							//flag_outputstage_2_d[0] = 1;

						}
						else
						{
							if(n1 == 1)
							{
								k_aux = k-((Dip*P)*floorf(k/(Dip*P)));
								k2 = floorf(k_aux/Dip);
								k1 = k_aux-(Dip*k2);
								X[k] = z[(k1*Dop*P)+(0*P)+ (k2%P)];
							}
							a = floorf(k/(Dip*P));
							X[k] = hipCaddf(X[k],hipCmulf(z[(k1*Dop*P)+(n1*P)+ (k2%P)],W[((n1*(k2+P*(a))*Dip)%N)-1]));

							///Flag
							//flag_outputstage_2_d[0] = 1;

						}

					}
					
					
					
					else
					{
						//Usando el m�todo filtering 2BF
						//printf("\n--- Caso (Filtro 2BF) ---\n");
						if((Dop-2) >= 1)
						{
							if(n1 == 0) 
							{
								k_aux = k-((Dip*P)*floorf(k/(Dip*P)));
								k2 = floorf(k_aux/Dip);
								k1 = k_aux-(Dip*k2);
								t1 = z[(k1*Dop*P)+((Dop-1)*P)+ (k2%P)];
								b = floorf(k/(Dip*P));
								t4 = hipCmulf(t1,make_hipFloatComplex(2*hipCrealf(W[(((k2+P*(b))*Dip)%N)-1]),0.0));

								///Flag
								//flag_outputstage_3_d[0] = 1;
							}

							if((n1 >= 1) && (n1 <= (Dop-2)))
							{
								t2 = t1;
								t1 = hipCaddf(z[(k1*Dop*P)+((-(n1-(Dop-1)))*P)+ (k2%P)],t4); 
								t3 = hipCmulf(t1,make_hipFloatComplex(2*hipCrealf(W[(((k2+P*(b))*Dip)%N)-1]),0.0));
								t4 = hipCsubf(t3,t2);
							}

							if(n1 == (Dop-1))
							{
								t5 = hipCaddf(z[(k1*Dop*P)+(0*P)+ (k2%P)],t4);
								X[k] = hipCsubf(t5,hipCmulf(t1,hipConjf(W[(((k2+P*(b))*Dip)%N)-1])));
							}

						}

						else
						{
							if(Dop == 1)
							{
								k_aux = k-((Dip*P)*floorf(k/(Dip*P)));
								k2 = floorf(k_aux/Dip);
								k1 = k_aux-(Dip*k2);
								t1 = z[(k1*Dop*P)+((Dop-1)*P)+ (k2%P)];
								X[k] = t1;

								///Flag
								//flag_outputstage_3_d[0] = 1;
							}
							else
							{
								k_aux = k-((Dip*P)*floorf(k/(Dip*P)));
								k2 = floorf(k_aux/Dip);
								k1 = k_aux-(Dip*k2);
								t1 = z[(k1*Dop*P)+((Dop-1)*P)+ (k2%P)];
								b = floorf(k/(Dip*P));
								t4 = hipCmulf(t1,make_hipFloatComplex(2*hipCrealf(W[(((k2+P*(b))*Dip)%N)-1]),0.0));
								t5 = hipCaddf(z[(k1*Dop*P)+(0*P)+ (k2%P)],t4);
								X[k] = hipCsubf(t5,hipCmulf(t1,hipConjf(W[(((k2+P*(b))*Dip)%N)-1])));

								///Flag
								//flag_outputstage_3_d[0] = 1;
							}

						}
						
					}
					
					

				}
				

			}
				
		}
	}
}


