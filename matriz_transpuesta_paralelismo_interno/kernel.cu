#include "hip/hip_runtime.h"
///VERSION PARALELISMO INTERNO
///Este programa encuentra la matriz transpuesta de "A" y el resultado lo guarda en la matriz "B"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <hip/hip_math_constants.h>
#include <iostream>

//2D surfaces
surface<void, cudaSurfaceType2D> A_surface;
surface<void, cudaSurfaceType2D> B_surface;

//Kernel
__global__ void MatrixTransposeKernel(int columns,int j)
{
	int data = 0;
	int i;
	i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < columns)
	{
		//A_host[r + (k*N)]=rand()%10;
		surf2Dread(&data,A_surface,i*4,j);
		surf2Dwrite(data,B_surface,j*4,i);	
	}

}

int main()
{
	int i,j,loop;
	unsigned long N,M,k,r;
	float suma;

	//Numero de iteraciones
	loop = 100;

	//N�mero de muestras
	const int muestras = 14;
	
	float promedio[muestras];

	///Se crean los archivos binarios donde se guardar�n los datos
    FILE *da;
    da = fopen("Matriz_transpuesta_paralelismo_interno.bin","a+b"); //Crea o sobre escribe archivo

	//Pausa
	printf("\n---PRESIONA UNA TECLA PARA CONTINUAR---\n\n");
	getchar();

	N=1;
	for(i=1;i<=muestras;i++)
    {
        ///N�mero de columnas de la matriz "A"
        N=N*2;

		///N�mero de renglones de la matriz "A"
		M=N;
    
        suma=0.0;
        for(j=0;j<loop;j++)
        {
			//Comandos necesarios para medir el tiempo
			float elapsedTime_app;
			hipEvent_t start_app, stop_app;
			hipEventCreate(&start_app);
			hipEventCreate(&stop_app);

			//Declaraci�n de variables
            int* A_host;
            int* B_host;

            //Se reserva memoria en host y device para las matrices A y B
            A_host = (int*) malloc(sizeof(int)*N*M);
            B_host = (int*) malloc(sizeof(int)*N*M);
           
			//Se dan valores a la matriz "A"
			for(k=0;k<M;k++)
            {
				for(r=0;r<N;r++)
				{
					A_host[r + (k*N)]=rand()%10;
				}
            }

			/*
			//Se imprime la matriz "A"
			printf("\n\n---MATRIZ A---\n\n");
			for(k=0;k<M;k++)
            {
				for(r=0;r<N;r++)
				{
					printf(" %d ",A_host[r + (k*N)]);
				}
				printf("\n");
            }
			printf("\n");
			*/

			//Allocate CUDA arrays in device memory
			hipChannelFormatDesc channelDesc;
			channelDesc = hipCreateChannelDesc<int>();
			hipArray* A_array;
			hipArray* B_array;
			hipMallocArray(&A_array,&channelDesc,N,M,hipArraySurfaceLoadStore);
			hipMallocArray(&B_array,&channelDesc,N,M,hipArraySurfaceLoadStore);

			//Copy to device memory some data located at address A_host in host memory
			hipMemcpyToArray(A_array,0,0,A_host,N*M*sizeof(int),hipMemcpyHostToDevice);

			//Bind the arrays to the surface references
			cudaBindSurfaceToArray(A_surface,A_array);
			cudaBindSurfaceToArray(B_surface,B_array);

			//Dimensionamiento del grid para la funci�n kernel
			//Dimensionamiento del Grid
			dim3 gridDim;
			gridDim.y = 1;
			gridDim.z = 1;
			//Dimensionamiento del block
			dim3 blockDim;
			blockDim.y = 1;
			blockDim.z = 1;

			if(N < 1024)
			{
				blockDim.x = N;
				gridDim.x = 1;
			}
			else
			{
				blockDim.x = 1024;
				gridDim.x = ceil(N/blockDim.x);
			}

			//---------------------------------------------------------------------------------------------
			//Se empieza a medir el tiempo de ejecucion de la aplicacion
			hipEventRecord(start_app,0);
			
			//Lanzamiento del kernel
			for(k=0; k < M; k++)
			{
				MatrixTransposeKernel<<<gridDim,blockDim >>>(N,k);
				hipDeviceSynchronize();
			}


			//---------------------------------------------------------------------------------------------
			//Comandos necesarios para medir el tiempo de la aplicacion (app)
			hipEventRecord(stop_app,0);
			hipEventSynchronize(stop_app);
			hipEventElapsedTime(&elapsedTime_app,start_app,stop_app);

			//Se leen los resultados de la GPU
			hipMemcpyFromArray(B_host,B_array,0,0,M*N*sizeof(int),hipMemcpyDeviceToHost);

			/*
			//Se imprime la matriz "B"
			printf("\n\n---MATRIZ B---\n\n");
			for(k=0;k<M;k++)
            {
				for(r=0;r<N;r++)
				{
					printf(" %d ",B_host[r + (k*N)]);
				}
				printf("\n");
            }
			printf("\n");
			*/

			//Se liberan memorias del Host y Device
			free(A_host);
			free(B_host);
			hipFreeArray(A_array);
			hipFreeArray(B_array);

			//Suma de todos los tiempos
			suma = suma + elapsedTime_app;

			//Se destruyen los eventos que miden el tiempo de la aplicacion
			hipEventDestroy(start_app);
			hipEventDestroy(stop_app);
		}
        promedio[i-1] = suma/(float)loop;
        printf("%d - Tiempo promedio para N = %ld >>> %f mS\n",i,N,promedio[i-1]);

    }
    fwrite(promedio,sizeof(float),muestras,da);
    fclose(da);
}

