/* Este programa calcula la versi�n paralela de la libreria cuFFT (sin podado) para N= 2^20*/
/// (20/09/2016)
///�sta versi�n sirve para graficar en matlab los tiempos de ejecuci�n (RADIX-2) SIN PODAR N= 2^20
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <hip/hip_math_constants.h>
#include <iostream>
#include <time.h>

//////////////////////////////////////////////////////////////////////////
///////////////////////DECLARACI�N DE FUNCIONES///////////////////////////
//////////////////////////////////////////////////////////////////////////
void vector_entrada_xn(int N);
void cuFFT_noprun(void);

//////////////////////////////////////////////////////////////////////////
/////////////////////DECLARACI�N DE VARIABLES GLOBALES////////////////////
//////////////////////////////////////////////////////////////////////////
hipFloatComplex *x_host;
hipFloatComplex *X_host;
hipFloatComplex *x_device;
hipFloatComplex *X_device;
hipfftComplex *in,*out;
FILE *db_open,*dc_open;

int N;

//////////////////////////////////////////////////////////////////////////
//////////////////////////DATOS DE ENTRADA////////////////////////////////
//////////////////////////////////////////////////////////////////////////
/// N  >>> N�mero de elementos del vector de entrada
/// Li >>> N�mero de elementos de entrada diferentes de cero
/// Lo >>> N�mero de elementos de salida requeridos
/// loop >>> N�mero de iteraciones
/// muestras >>> N�mero de muestras

//////////////////////////////////////////////////////////////////////////
///////////////////////////DATOS DE SALIDA////////////////////////////////
//////////////////////////////////////////////////////////////////////////
/// X >>> Vector de salida

//////////////////////////////////////////////////////////////////////////
/////////////////// SE INGRESAN LOS DATOS DE ENTRADA /////////////////////
//////////////////////////////////////////////////////////////////////////

///Ingrese el n�mero de iteraciones requeridas
const int loop = 300;

///Ingrese el valor de N_max

const int N_max = 20;



//////////////////////////////////////////////////////////////////////////
//////////////////////////FUNCION PRINCIPAL///////////////////////////////
//////////////////////////////////////////////////////////////////////////

//Funci�n principal 
int main()
{
	int i,j,i_N,j_res,k_res,cont,i_prom;
	float suma;
	float promedio[1];

	FILE *da;
	da = fopen("Tiempos_cuFFT__noprun_N20.bin","a+b"); //Crea o sobre escribe archivo

    //Pausa
	printf("\n---PRESIONA UNA TECLA PARA CONTINUAR---\n\n");
	getchar();
	
	for(i_N = N_max;i_N <= N_max;i_N++)
    {
        N = (int )pow(2,i_N);
        printf("\n N = %d \n",N);

		///Se abre el archivo binario
		db_open = fopen("Entrada_real_N20_C.bin","rb");
        dc_open = fopen("Entrada_imag_N20_C.bin","rb");

		suma=0.0;
		for(j=0;j<loop;j++)
		{
		
			//Comandos necesarios para medir el tiempo
			float elapsedTime_app;
			hipEvent_t start_app, stop_app;
			hipEventCreate(&start_app);
			hipEventCreate(&stop_app);

			//Se generan en el host los valores del vector de entrada x[n] 
			vector_entrada_xn(N);

			//---------------------------------------------------------------------------------------------
			//Se empieza a medir el tiempo de ejecucion de la aplicacion
			hipEventRecord(start_app,0);

			//Funci�n auxiliar del host para ejecutar la etapa intermedia
			cuFFT_noprun();

			//---------------------------------------------------------------------------------------------
			//Comandos necesarios para medir el tiempo de la aplicacion (app)
			hipEventRecord(stop_app,0);
			hipEventSynchronize(stop_app);
			hipEventElapsedTime(&elapsedTime_app,start_app,stop_app);

			//Suma de todos los tiempos
			suma = suma + elapsedTime_app;

			//Se destruyen los eventos que miden el tiempo de la aplicacion
			hipEventDestroy(start_app);
			hipEventDestroy(stop_app);

			//Se liberan memorias del Host y Device
			free(x_host);
			free(X_host);
			hipFree(x_device);
			hipFree(X_device);
			

		}

		
		promedio[0] = suma/(float)loop;
		fclose(db_open);
        fclose(dc_open);
			
			
		
	}
	fwrite(promedio,sizeof(float),1,da);
	printf("\n\n- Tiempo promedio para N = %d >>> %f mS\n",N,promedio[0]);
    fclose(da);				

	return EXIT_SUCCESS;
}

//////////////////////////////////////////////////////////////////////////
/////////////////////////FUNCIONES SECUNDARIAS////////////////////////////
//////////////////////////////////////////////////////////////////////////

//�sta funci�n genera el vector de entrada x[n]
void vector_entrada_xn(int N)
{
	//Declaraci�n de variables locales
	int k;
	float *buffer_real,*buffer_imag;

	//Se reserva memoria para xn_host en el host
	x_host = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*N);

	//Se reserva memoria para "X" en el host
	X_host = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*N);

	buffer_real = (float*)malloc(sizeof(float)*(pow(2,N_max)));
	buffer_imag = (float*)malloc(sizeof(float)*(pow(2,N_max)));

	///Se lee el vector de entrada del archivo binario
	fread(buffer_real,sizeof(float),(int)pow(2,N_max),db_open);
    fread(buffer_imag,sizeof(float),(int)pow(2,N_max),dc_open);


	//Se dan valores a x[n]
	for(k = 0;k < N; k++)
	{
		//x_host[k] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%11));
		//x_host[k] = make_cuFloatComplex((float)(k + 1),(float)(0.0));
		x_host[k] = make_hipFloatComplex(buffer_real[k],buffer_imag[k]);
	}

	/*
	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(k=0;k<N;k++) 
	{
		printf(" %d-> (%f) + (%f)\n",k+1,cuCrealf(x_host[k]),cuCimagf(x_host[k]));
	}
	*/
	
	free(buffer_real);
	free(buffer_imag);
}


//Funci�n auxiliar del host para calcular la etapa intermedia en el device
void cuFFT_noprun(void)
{
	//////////////////////////////////////////////////////////////////////////
	////////////////////////////ETAPA INTERMEDIA//////////////////////////////
	//////////////////////////////////////////////////////////////////////////

	//Declaraci�n de variables locales
	int k1,k2,n1,m;
	int n[1] = {N};
	int inembed[1] = {NULL};
	int onembed[1] = {NULL};

	//Asignaci�n de memoria en el device para el arreglo "x_device"
	hipMalloc((void**)&x_device,N*sizeof(hipFloatComplex));
	
	//Asignaci�n de memoria en el device para "X"
	hipMalloc((void**)&X_device,N*sizeof(hipFloatComplex));

	//Se pasa el arreglo x_host a x_device
	hipMemcpy(x_device,x_host,N*sizeof(hipFloatComplex),hipMemcpyHostToDevice);

	//Asignaci�n de memoria en el device para "in" y "out"
	hipMalloc((void**)&in,sizeof(hipfftComplex)*N);
	hipMalloc((void**)&out,sizeof(hipfftComplex)*N);

	//Se copia el arreglo "x_device" al arreglo "in"
	hipMemcpy(in,x_device,sizeof(hipFloatComplex)*N,hipMemcpyDeviceToDevice);

	//Se crea un plan
	hipfftHandle plan;
	hipfftPlanMany(&plan,1,n,inembed,1,1,onembed,1,1,HIPFFT_C2C,1);

	//Ejecuci�n del plan
	hipfftExecC2C(plan,in,out,HIPFFT_FORWARD);

	//Esperar que el kernel termine de ejecutarse totalmente
	hipDeviceSynchronize();

	//Se copian los datos del arreglo "out" al arreglo "z_device"
	hipMemcpy(X_device,out,sizeof(hipfftComplex)*N,hipMemcpyDeviceToDevice);

	//Copia del arreglo "X" del device hacia el host
	hipMemcpy(X_host,X_device,sizeof(hipFloatComplex)*N,hipMemcpyDeviceToHost);

	//Se destruye el plan
	hipfftDestroy(plan);

	//Se liberan los arreglos "in" y "out"
	hipFree(in);
	hipFree(out);

	/*
	//Se imprimen los valores de "X_host"
	///Imprimir X[k]
	printf("\n\n--- ARREGLO X[k] ---\n\n");
	for(m=0;m<=N-1;m++)
	{
		printf("\n X[%d] = %.4f + (%.4f)",m,cuCrealf(X_host[m]),cuCimagf(X_host[m]));
		//fprintf(da,"%.4f %.4f\n",creal(X[i]),cimag(X[i]));
	}
	*/
}

