//Calculo de la FFT 3D usando "cufftPlanMany"

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>

#define RENGLONES 2
#define COLUMNAS 3
#define PROFUNDIDAD 4


int main()
{
	int i,j,k;
	int n[3] = {RENGLONES,COLUMNAS,PROFUNDIDAD};
	int inembed[3] = {RENGLONES,COLUMNAS,PROFUNDIDAD};
	int onembed[3] = {RENGLONES,COLUMNAS,PROFUNDIDAD};
	hipFloatComplex *h_xn;
	hipFloatComplex *h_xn_trans;
	hipFloatComplex *h_Xk;
	hipFloatComplex *h_Xk_trans;
	hipfftComplex *in,*out;

	//Se reserva memoria para h_xn en el host
	h_xn = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para h_xn_trans en el host
	h_xn_trans = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para h_Xk en el host
	h_Xk = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para h_Xk_trans en el host
	h_Xk_trans = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se dan valores a x[n]
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				//h_xn[i] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%21));
				h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j] = make_hipFloatComplex((float)(((k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j) + 1),(float)(0.0));
			}
			
		}
	}

	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f) ",hipCrealf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}

	//Se saca la transpuesta del arreglo tridimensional "h_xn"
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				h_xn_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j] = make_hipFloatComplex(hipCrealf(h_xn[(k*RENGLONES*COLUMNAS)+(j*COLUMNAS)+i]),hipCimagf(h_xn[(k*RENGLONES*COLUMNAS)+(j*COLUMNAS)+i]));
			}
			
		}
	}

	/*
	//Se imprimen los valores de entrada x[n] (matriz transpuesta)
	printf("\n---ELEMENTOS DE ENTRADA x[n] (Matriz transpuesta)---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				printf(" (%f) + (%f) ",cuCrealf(h_xn_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]),cuCimagf(h_xn_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}
	*/


	//Se reserva memoria para "in" en el device
	hipMalloc((void**)&in,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para "out" en el device
	hipMalloc((void**)&out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se copian los datos de h_xn_trans >>> in 
	hipMemcpy(in,h_xn_trans,sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD,hipMemcpyHostToDevice);

	//CUFFT plan
	hipfftHandle plan;
	hipfftPlanMany(&plan,1,n,inembed,1,RENGLONES,onembed,1,RENGLONES,HIPFFT_C2C,COLUMNAS*PROFUNDIDAD);

	//Ejecucion de la fft
	hipfftExecC2C(plan,in,out,HIPFFT_FORWARD);

	//Se copian los datos de out >>> h_Xk
	hipMemcpy(h_Xk_trans,out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD,hipMemcpyDeviceToHost);

	/*
	//Se imprimen los valores de salida X[k] (Matriz transpuesta h_Xk_trans)
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				printf(" (%f) + (%f) ",cuCrealf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]),cuCimagf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}
	*/

	//Se saca la transpuesta del arreglo tridimensional "h_Xk_trans"
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				h_Xk[(k*RENGLONES*COLUMNAS)+(j*COLUMNAS)+i] = make_hipFloatComplex(hipCrealf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]),hipCimagf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]));
			}
			
		}
	}

	//Se imprimen los valores de salida X[k] 
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f) ",hipCrealf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}



	//Se destruye el plan
	hipfftDestroy(plan);

	//Se liberan memorias
	free(h_xn);
	free(h_Xk);
	free(h_xn_trans);
	free(h_Xk_trans);
	hipFree(in);
	hipFree(out);
}
