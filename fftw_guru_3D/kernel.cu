//Calculo de la FFT 3D usando "fftw_plan_guru_dft"----->>>>sin transpuesta

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>


#define RENGLONES 2
#define COLUMNAS 3
#define PROFUNDIDAD 2


int main()
{
	int i,j,k;
	//int n[1] = {RENGLONES};
	//int n[3] = {PROFUNDIDAD,COLUMNAS,RENGLONES};
	//int inembed[1] = {RENGLONES};
	//int inembed[3] = {PROFUNDIDAD,COLUMNAS,RENGLONES};
	//int onembed[1] = {RENGLONES};
	//int onembed[3] = {PROFUNDIDAD,COLUMNAS,RENGLONES};
	
	fftwf_iodim dims[1];
	fftwf_iodim howmany_dims[1];

	dims[0].n = 1;
	//dims[1].n = COLUMNAS;
	dims[0].is = 1;
	//dims[1].is = 1;
	dims[0].os = 1;
	//dims[1].os = 1;

	howmany_dims[0].n = 3;
	//howmany_dims[1].n = 2;
	//howmany_dims[1].n = RENGLONES*COLUMNAS;
	howmany_dims[0].is =4;
	//howmany_dims[1].is =3;
	//howmany_dims[1].is =COLUMNAS;
	howmany_dims[0].os =4;
	//howmany_dims[1].os =3;
	//howmany_dims[1].os =COLUMNAS;

	
	
	hipFloatComplex *h_xn;
	//cuFloatComplex *h_xn_trans;
	hipFloatComplex *h_Xk;
	//cuFloatComplex *h_Xk_trans;
	fftwf_complex *in,*out;
	//cufftComplex *in,*out;

	//Se reserva memoria para h_xn en el host
	h_xn = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para h_xn_trans en el host
	//h_xn_trans = (cuFloatComplex*)malloc(sizeof(cuFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para h_Xk en el host
	h_Xk = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para h_Xk_trans en el host
	//h_Xk_trans = (cuFloatComplex*)malloc(sizeof(cuFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se dan valores a x[n]
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				//h_xn[i] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%21));
				h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j] = make_hipFloatComplex((float)(((k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j) + 1),(float)(0.0));
			}
			
		}
	}

	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f) ",hipCrealf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}
	/*
	//Se saca la transpuesta del arreglo tridimensional "h_xn"
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				h_xn_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j] = make_cuFloatComplex(cuCrealf(h_xn[(k*RENGLONES*COLUMNAS)+(j*COLUMNAS)+i]),cuCimagf(h_xn[(k*RENGLONES*COLUMNAS)+(j*COLUMNAS)+i]));
			}
			
		}
	}
	*/
	/*
	//Se imprimen los valores de entrada x[n] (matriz transpuesta)
	printf("\n---ELEMENTOS DE ENTRADA x[n] (Matriz transpuesta)---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				printf(" (%f) + (%f) ",cuCrealf(h_xn_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]),cuCimagf(h_xn_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}
	*/


	//Se reserva memoria para "in" en el device
	hipMalloc((void**)&in,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para "out" en el device
	hipMalloc((void**)&out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se copian los datos de h_xn >>> in 
	hipMemcpy(in,h_xn,sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD,hipMemcpyHostToDevice);

	//CUFFT plan
	fftwf_plan plan;
	//cufftHandle plan;
	//cufftPlanMany(&plan,1,n,inembed,COLUMNAS,1,onembed,COLUMNAS,1,CUFFT_C2C,COLUMNAS*PROFUNDIDAD);
	//plan = fftwf_plan_dft(1,n,in,out,FFTW_FORWARD,FFTW_ESTIMATE);
	plan = fftwf_plan_guru_dft(1,dims,1,howmany_dims,in,out,FFTW_FORWARD,FFTW_ESTIMATE);
	
	//Ejecucion de la fft
	fftwf_execute(plan);
	//cufftExecC2C(plan,in,out,CUFFT_FORWARD);
	//fftwf_execute_dft(plan,in,out);
	//Se copian los datos de out >>> h_Xk
	hipMemcpy(h_Xk,out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD,hipMemcpyDeviceToHost);

	/*
	//Se imprimen los valores de salida X[k] (Matriz transpuesta h_Xk_trans)
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				printf(" (%f) + (%f) ",cuCrealf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]),cuCimagf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}
	*/
	/*
	//Se saca la transpuesta del arreglo tridimensional "h_Xk_trans"
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				h_Xk[(k*RENGLONES*COLUMNAS)+(j*COLUMNAS)+i] = make_cuFloatComplex(cuCrealf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]),cuCimagf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]));
			}
			
		}
	}
	*/
	//Se imprimen los valores de salida X[k] 
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f) ",hipCrealf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}



	//Se destruye el plan
	fftwf_destroy_plan(plan);
	//cufftDestroy(plan);

	//Se liberan memorias
	free(h_xn);
	free(h_Xk);
	//free(h_xn_trans);
	//free(h_Xk_trans);
	hipFree(in);
	hipFree(out);
}