#include "hip/hip_runtime.h"
//Calculo de la FFT 3D usando "cufftw_Plan_Many_dft" y paralelismo dinamico

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>

#define RENGLONES 2
#define COLUMNAS 3
#define PROFUNDIDAD 2


__global__ void kernel_padre(hipfftComplex *in, hipfftComplex *out)
{
	int n[1] = {RENGLONES};
	int inembed[1] = {RENGLONES};
	int onembed[1] = {RENGLONES};
	
	
	//CUFFT plan
	//fftwf_plan plan;
	hipfftHandle plan;
	hipfftPlanMany(&plan,1,n,inembed,1,RENGLONES,onembed,1,RENGLONES,HIPFFT_C2C,COLUMNAS*PROFUNDIDAD);
	//plan = fftwf_plan_many_dft(1,n,COLUMNAS*PROFUNDIDAD,in,inembed,1,RENGLONES,out,onembed,1,RENGLONES,FFTW_FORWARD,FFTW_ESTIMATE);

	//Ejecucion de la fft
	//fftwf_execute(plan);
	hipfftExecC2C(plan,in,out,HIPFFT_FORWARD);

	//Se destruye el plan
	//fftwf_destroy_plan(plan);
	hipfftDestroy(plan);

}





int main()
{
	int i,j,k;
	//int n[1] = {RENGLONES};
	//int inembed[1] = {RENGLONES};
	//int onembed[1] = {RENGLONES};
	hipFloatComplex *h_xn;
	hipFloatComplex *h_xn_trans;
	hipFloatComplex *h_Xk;
	hipFloatComplex *h_Xk_trans;
	//fftwf_complex *in,*out;
	hipfftComplex *in,*out;

	//Se reserva memoria para h_xn en el host
	h_xn = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para h_xn_trans en el host
	h_xn_trans = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para h_Xk en el host
	h_Xk = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para h_Xk_trans en el host
	h_Xk_trans = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se dan valores a x[n]
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				//h_xn[i] = make_hipFloatComplex((float)(rand()%11),(float)(rand()%21));
				h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j] = make_hipFloatComplex((float)(((k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j) + 1),(float)(0.0));
			}
			
		}
	}

	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f) ",hipCrealf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}

	//Se saca la transpuesta del arreglo tridimensional "h_xn"
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				h_xn_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j] = make_hipFloatComplex(hipCrealf(h_xn[(k*RENGLONES*COLUMNAS)+(j*COLUMNAS)+i]),hipCimagf(h_xn[(k*RENGLONES*COLUMNAS)+(j*COLUMNAS)+i]));
			}
			
		}
	}

	/*
	//Se imprimen los valores de entrada x[n] (matriz transpuesta)
	printf("\n---ELEMENTOS DE ENTRADA x[n] (Matriz transpuesta)---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				printf(" (%f) + (%f) ",hipCrealf(h_xn_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]),hipCimagf(h_xn_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}
	*/


	//Se reserva memoria para "in" en el device
	hipMalloc((void**)&in,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se reserva memoria para "out" en el device
	hipMalloc((void**)&out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD);

	//Se copian los datos de h_xn_trans >>> in 
	hipMemcpy(in,h_xn_trans,sizeof(hipFloatComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD,hipMemcpyHostToDevice);

	dim3 dimBlock(1,1,1);
	dim3 dimGrid(1,1,1);
	kernel_padre<<<dimGrid,dimBlock>>> (in,out);

	//CUFFT plan
	//fftwf_plan plan;
	//hipfftPlanMany(&plan,1,n,inembed,1,RENGLONES,onembed,1,RENGLONES,HIPFFT_C2C,COLUMNAS*PROFUNDIDAD);
	//plan = fftwf_plan_many_dft(1,n,COLUMNAS*PROFUNDIDAD,in,inembed,1,RENGLONES,out,onembed,1,RENGLONES,FFTW_FORWARD,FFTW_ESTIMATE);

	//Ejecucion de la fft
	//fftwf_execute(plan);

	//Se copian los datos de out >>> h_Xk
	hipMemcpy(h_Xk_trans,out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD,hipMemcpyDeviceToHost);

	/*
	//Se imprimen los valores de salida X[k] (Matriz transpuesta h_Xk_trans)
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				printf(" (%f) + (%f) ",hipCrealf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]),hipCimagf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}
	*/

	//Se saca la transpuesta del arreglo tridimensional "h_Xk_trans"
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<COLUMNAS;i++)
		{
			for(j=0;j<RENGLONES;j++)
			{
				h_Xk[(k*RENGLONES*COLUMNAS)+(j*COLUMNAS)+i] = make_hipFloatComplex(hipCrealf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]),hipCimagf(h_Xk_trans[(k*RENGLONES*COLUMNAS)+(i*RENGLONES)+j]));
			}
			
		}
	}

	//Se imprimen los valores de salida X[k] 
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f) ",hipCrealf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");	
		}
		printf("\n\n");
	}



	//Se destruye el plan
	//fftwf_destroy_plan(plan);

	//Se liberan memorias
	free(h_xn);
	free(h_Xk);
	free(h_xn_trans);
	free(h_Xk_trans);
	hipFree(in);
	hipFree(out);
}


