//Calculo de la FFT 3D utilizando la funcion cufftPlan3D();

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>


#define RENGLONES 3
#define COLUMNAS 3
#define PROFUNDIDAD 2


int main()
{
	int i,j,k;
	hipFloatComplex *h_xn;
	hipFloatComplex *h_Xk;
	hipfftComplex *in,*out;

	//Se reserva memoria para h_xn en el host
	h_xn = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD);

	//Se reserva memoria para h_Xk en el host
	h_Xk = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD);

	//Se dan valores a x[n]
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				//h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%21));
				h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j] = make_hipFloatComplex((float)(((k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j) + 1),(float)(0.0));
			}
		}
	}
	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f)",hipCrealf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");
		}
		printf("\n\n");
	}
	//Se reserva memoria para "in" en el device
	hipMalloc((void**)&in,sizeof(hipfftComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD);

	//Se reserva memoria para "out" en el device
	hipMalloc((void**)&out,sizeof(hipfftComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD);

	//Se copian los datos de h_xn >>> in 
	hipMemcpy(in,h_xn,sizeof(hipFloatComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD,hipMemcpyHostToDevice);

	//CUFFT plan
	hipfftHandle plan;
	hipfftPlan3d(&plan,PROFUNDIDAD,RENGLONES,COLUMNAS, HIPFFT_C2C);

	

	//Ejecucion de la fft
	hipfftExecC2C(plan,in,out,HIPFFT_FORWARD);

	//Se copian los datos de out >>> h_Xk
	hipMemcpy(h_Xk,out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD,hipMemcpyDeviceToHost);

	//Se imprimen los valores de salida X[k]
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f)",hipCrealf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");
		}
		printf("\n\n");
	}
	//Se destruye el plan
	hipfftDestroy(plan);

	//Se liberan memorias
	free(h_xn);
	free(h_Xk);
	hipFree(in);
	hipFree(out);
}