
//Calculo de la FFT 2D utilizando la funcion cufftPlan2D();

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>


#define RENGLONES 3
#define COLUMNAS 3


int main()
{
	int i,j;
	hipFloatComplex *h_xn;
	hipFloatComplex *h_Xk;
	hipfftComplex *in,*out;

	//Se reserva memoria para h_xn en el host
	h_xn = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*COLUMNAS*RENGLONES);

	//Se reserva memoria para h_Xk en el host
	h_Xk = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*COLUMNAS*RENGLONES);

	//Se dan valores a x[n]
	for(i=0;i<RENGLONES;i++)
	{
		for(j=0;j<COLUMNAS;j++)
		{
			//h_xn[(i*COLUMNAS)+j] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%21));
			h_xn[(i*COLUMNAS)+j] = make_hipFloatComplex((float)(((i*COLUMNAS)+j) + 1),(float)(0.0));
		}
	}

	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(i=0;i<RENGLONES;i++)
	{
		for(j=0;j<COLUMNAS;j++)
		{
			printf(" (%f) + (%f)",hipCrealf(h_xn[(i*COLUMNAS)+j]),hipCimagf(h_xn[(i*COLUMNAS)+j]));
		}
		printf("\n");
	}

	//Se reserva memoria para "in" en el device
	hipMalloc((void**)&in,sizeof(hipfftComplex)*COLUMNAS*RENGLONES);

	//Se reserva memoria para "out" en el device
	hipMalloc((void**)&out,sizeof(hipfftComplex)*COLUMNAS*RENGLONES);

	//Se copian los datos de h_xn >>> in 
	hipMemcpy(in,h_xn,sizeof(hipFloatComplex)*COLUMNAS*RENGLONES,hipMemcpyHostToDevice);

	//CUFFT plan
	hipfftHandle plan;
	hipfftPlan2d(&plan,RENGLONES,COLUMNAS, HIPFFT_C2C);

	

	//Ejecucion de la fft
	hipfftExecC2C(plan,in,out,HIPFFT_FORWARD);

	//Se copian los datos de out >>> h_Xk
	hipMemcpy(h_Xk,out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS,hipMemcpyDeviceToHost);

	//Se imprimen los valores de salida X[k]
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(i=0;i<RENGLONES;i++)
	{
		for(j=0;j<COLUMNAS;j++)
		{
			printf(" (%f) + (%f)",hipCrealf(h_Xk[(i*COLUMNAS)+j]),hipCimagf(h_Xk[(i*COLUMNAS)+j]));
		}
		printf("\n");
	}

	//Se destruye el plan
	hipfftDestroy(plan);

	//Se liberan memorias
	free(h_xn);
	free(h_Xk);
	hipFree(in);
	hipFree(out);
}

