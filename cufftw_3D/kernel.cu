//Calculo de la FFT 3D utilizando la funcion "fftwf_plan_dft_3d";

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>

#define RENGLONES 3
#define COLUMNAS 4
#define PROFUNDIDAD 2


int main()
{
	int i,j,k;
	hipFloatComplex *h_xn;
	hipFloatComplex *h_Xk;
	fftwf_complex *in,*out;

	//Se reserva memoria para h_xn en el host
	h_xn = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD);

	//Se reserva memoria para h_Xk en el host
	h_Xk = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD);

	//Se dan valores a x[n]
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				//h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%21));
				h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j] = make_hipFloatComplex((float)(((k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j) + 1),(float)(0.0));
			}
		}
	}
	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f)",hipCrealf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_xn[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");
		}
		printf("\n\n");
	}
	//Se reserva memoria para "in" en el device
	hipMalloc((void**)&in,sizeof(hipfftComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD);

	//Se reserva memoria para "out" en el device
	hipMalloc((void**)&out,sizeof(hipfftComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD);

	//Se copian los datos de h_xn >>> in 
	hipMemcpy(in,h_xn,sizeof(hipFloatComplex)*COLUMNAS*RENGLONES*PROFUNDIDAD,hipMemcpyHostToDevice);

	//CUFFT plan
	fftwf_plan plan;
	plan = fftwf_plan_dft_3d(PROFUNDIDAD,RENGLONES,COLUMNAS,in,out,FFTW_FORWARD,FFTW_ESTIMATE);

	

	//Ejecucion de la fft
	fftwf_execute(plan);

	//Se copian los datos de out >>> h_Xk
	hipMemcpy(h_Xk,out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS*PROFUNDIDAD,hipMemcpyDeviceToHost);

	//Se imprimen los valores de salida X[k]
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(k=0;k<PROFUNDIDAD;k++)
	{
		for(i=0;i<RENGLONES;i++)
		{
			for(j=0;j<COLUMNAS;j++)
			{
				printf(" (%f) + (%f)",hipCrealf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]),hipCimagf(h_Xk[(k*RENGLONES*COLUMNAS)+(i*COLUMNAS)+j]));
			}
			printf("\n");
		}
		printf("\n\n");
	}
	//Se destruye el plan
	fftwf_destroy_plan(plan);

	//Se liberan memorias
	free(h_xn);
	free(h_Xk);
	hipFree(in);
	hipFree(out);
}
