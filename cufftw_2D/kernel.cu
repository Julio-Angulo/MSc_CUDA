//Calculo de la FFT 2D utilizando la funcion "fftwf_plan_dft_2d";

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>


#define RENGLONES 3
#define COLUMNAS 4


int main()
{
	int i,j;
	hipFloatComplex *h_xn;
	hipFloatComplex *h_Xk;
	fftwf_complex  *in,*out;

	//Se reserva memoria para h_xn en el host
	h_xn = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*COLUMNAS*RENGLONES);

	//Se reserva memoria para h_Xk en el host
	h_Xk = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*COLUMNAS*RENGLONES);

	//Se dan valores a x[n]
	for(i=0;i<RENGLONES;i++)
	{
		for(j=0;j<COLUMNAS;j++)
		{
			//h_xn[(i*COLUMNAS)+j] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%21));
			h_xn[(i*COLUMNAS)+j] = make_hipFloatComplex((float)(((i*COLUMNAS)+j) + 1),(float)(0.0));
		}
	}

	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(i=0;i<RENGLONES;i++)
	{
		for(j=0;j<COLUMNAS;j++)
		{
			printf(" (%f) + (%f)",hipCrealf(h_xn[(i*COLUMNAS)+j]),hipCimagf(h_xn[(i*COLUMNAS)+j]));
		}
		printf("\n");
	}

	//Se reserva memoria para "in" en el device
	hipMalloc((void**)&in,sizeof(hipfftComplex)*COLUMNAS*RENGLONES);

	//Se reserva memoria para "out" en el device
	hipMalloc((void**)&out,sizeof(hipfftComplex)*COLUMNAS*RENGLONES);

	//Se copian los datos de h_xn >>> in 
	hipMemcpy(in,h_xn,sizeof(hipFloatComplex)*COLUMNAS*RENGLONES,hipMemcpyHostToDevice);

	//CUFFT plan
	fftwf_plan plan;
	plan = fftwf_plan_dft_2d(RENGLONES,COLUMNAS,in,out,FFTW_FORWARD,FFTW_ESTIMATE);

	

	//Ejecucion de la fft
	fftwf_execute(plan);

	//Se copian los datos de out >>> h_Xk
	hipMemcpy(h_Xk,out,sizeof(hipfftComplex)*RENGLONES*COLUMNAS,hipMemcpyDeviceToHost);

	//Se imprimen los valores de salida X[k]
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(i=0;i<RENGLONES;i++)
	{
		for(j=0;j<COLUMNAS;j++)
		{
			printf(" (%f) + (%f)",hipCrealf(h_Xk[(i*COLUMNAS)+j]),hipCimagf(h_Xk[(i*COLUMNAS)+j]));
		}
		printf("\n");
	}

	//Se destruye el plan
	fftwf_destroy_plan(plan);

	//Se liberan memorias
	free(h_xn);
	free(h_Xk);
	hipFree(in);
	hipFree(out);
}


