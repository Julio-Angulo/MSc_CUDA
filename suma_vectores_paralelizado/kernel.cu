#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <hip/hip_math_constants.h>
#include <iostream>

__global__ void VectorAddKernel(int *a, int *b, int *c, int N)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int z = blockDim.z * blockIdx.z + threadIdx.z;
	int tid = x + (y * blockDim.x * gridDim.x) + (z * blockDim.x * gridDim.x * blockDim.y * gridDim.y);
	if (tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}

}

int main()
{
	int i,j,loop,muestras;
	unsigned long N,k;
	float suma;

	//Numero de iteraciones
	loop = 500;

	//N�mero de muestras
    muestras=130;
	
	float promedio[130];

	///Se crean los archivos binarios donde se guardar�n los datos
    FILE *da;
    da = fopen("Suma_vectores_500_iteraciones_paralelizado.bin","a+b"); //Crea o sobre escribe archivo

	//Pausa
	printf("\n---PRESIONA UNA TECLA PARA CONTINUAR---\n\n");
	getchar();

	for(i=1;i<=muestras;i++)
    {
        ///N�mero de elementos de los vectores a,b y c
        N=2048*i;
    
        suma=0.0;
        for(j=0;j<loop;j++)
        {
			//Comandos necesarios para medir el tiempo
			float elapsedTime_app;
			hipEvent_t start_app, stop_app;
			hipEventCreate(&start_app);
			hipEventCreate(&stop_app);

			//Declaraci�n de variables
            int *a_host;
            int *b_host;
            int *c_host;
			int *a_device;
            int *b_device;
            int *c_device;
            
			//Se reserva memoria en host y device para a, b y c
            a_host = (int*) malloc(sizeof(int)*N);
            b_host = (int*) malloc(sizeof(int)*N);
            c_host = (int*) malloc(sizeof(int)*N);
			hipMalloc((void**)&a_device,sizeof(int)*N);
			hipMalloc((void**)&b_device,sizeof(int)*N);
			hipMalloc((void**)&c_device,sizeof(int)*N);

			//Se dan valores a los vectores a y b
            for(k=0;k<N;k++)
            {
                a_host[k]=rand()%11;
                b_host[k]=rand()%21;
            }

			//Dimensionamiento del grid para la funci�n kernel
			dim3 gridDim(1,1,1);
			dim3 blockDim(1,1,1);
			int maxNumThreads = 1024;
			dim3 maxNumBlocks(2147483647,65535,65535);
			if(N <= maxNumThreads)
			{
				blockDim.x = N;
			}
			else
			{
				blockDim.x = maxNumThreads;
				if(ceil(N/blockDim.x) <= maxNumBlocks.x)
				{
					gridDim.x = ceil(N/blockDim.x);
				}
				else
				{
					gridDim.x = maxNumBlocks.x;
					if(ceil(N/(blockDim.x*gridDim.x)) <= maxNumBlocks.y)
					{
						gridDim.y = ceil(N/(blockDim.x*gridDim.x));
					}
					else
					{
						gridDim.y = maxNumBlocks.y;
						if(ceil(N/(blockDim.x*gridDim.x*blockDim.y*gridDim.y)) <=maxNumBlocks.z)
						{
							gridDim.z = ceil(N/(blockDim.x*gridDim.x*blockDim.y*gridDim.y));
						}
						else
						{
							printf("El n�mero de datos excede la capacidad del grid");
						}
					}
				}
			}

			//Env�o de los arreglos a y b al device
			hipMemcpy(a_device,a_host,sizeof(int)*N,hipMemcpyHostToDevice);
			hipMemcpy(b_device,b_host,sizeof(int)*N,hipMemcpyHostToDevice);

			//---------------------------------------------------------------------------------------------
			//Se empieza a medir el tiempo de ejecucion de la aplicacion
			hipEventRecord(start_app,0);

			//Lanzamiento del kernel
			VectorAddKernel<<<gridDim,blockDim>>>(a_device,b_device,c_device,N);
			hipDeviceSynchronize();

			//------------------------------------------------------------------------------------------
			//Comandos necesarios para medir el tiempo de la aplicacion (app)
			hipEventRecord(stop_app,0);
			hipEventSynchronize(stop_app);
			hipEventElapsedTime(&elapsedTime_app,start_app,stop_app);

			//Se leen los resultados de la GPU
			hipMemcpy(c_host,c_device,sizeof(int)*N,hipMemcpyDeviceToHost);

			//Se liberan memorias del Host y Device
			free(a_host);
			free(b_host);
			free(c_host);
			hipFree(a_device);
			hipFree(b_device);
			hipFree(c_device);

			//Suma de todos los tiempos
			suma = suma + elapsedTime_app;

			//Se destruyen los eventos que miden el tiempo de la aplicacion
			hipEventDestroy(start_app);
			hipEventDestroy(stop_app);
		}
        promedio[i-1] = suma/(float)loop;
        printf("%d - Tiempo promedio para N = %ld >>> %f mS\n",i,N,promedio[i-1]);

    }
    fwrite(promedio,sizeof(float),muestras,da);
    fclose(da);
}
