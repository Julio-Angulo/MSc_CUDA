//Calculo de la FFT 1D usando "cufftPlan1d".

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>
#include <hipfft/hipfftXt.h>

#define SIGNAL_SIZE 10



int main()
{
	int i;
	hipFloatComplex *h_xn;
	hipFloatComplex *h_Xk;
	hipfftComplex *in,*out;

	//Se reserva memoria para h_xn en el host
	h_xn = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*SIGNAL_SIZE);

	//Se reserva memoria para h_Xk en el host
	h_Xk = (hipFloatComplex*)malloc(sizeof(hipFloatComplex)*SIGNAL_SIZE);

	//Se dan valores a x[n]
	for(i=0;i<SIGNAL_SIZE;i++)
	{
		
			//h_xn[i] = make_cuFloatComplex((float)(rand()%11),(float)(rand()%21));
			h_xn[i] = make_hipFloatComplex((float)(i+1),(float)(0.0));
	}

	//Se imprimen los valores de entrada x[n]
	printf("\n---ELEMENTOS DE ENTRADA x[n]---\n\n");
	for(i = 0; i<SIGNAL_SIZE;i++)
	{
		printf(" x[%d] = (%f) + (%f)\n",i,hipCrealf(h_xn[i]),hipCimagf(h_xn[i]));
	}

	//Se reserva memoria para "in" en el device
	hipMalloc((void**)&in,sizeof(hipfftComplex)*SIGNAL_SIZE);

	//Se reserva memoria para "out" en el device
	hipMalloc((void**)&out,sizeof(hipfftComplex)*SIGNAL_SIZE);

	//Se copian los datos de h_xn >>> in 
	hipMemcpy(in,h_xn,sizeof(hipFloatComplex)*SIGNAL_SIZE,hipMemcpyHostToDevice);

	//CUFFT plan
	hipfftHandle plan;
	hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_C2C, 1);

	//Ejecucion de la fft
	hipfftExecC2C(plan,in,out,HIPFFT_FORWARD);

	//Se copian los datos de out >>> h_Xk
	hipMemcpy(h_Xk,out,sizeof(hipfftComplex)*SIGNAL_SIZE,hipMemcpyDeviceToHost);

	//Se imprimen los valores de salida X[k]
	printf("\n---ELEMENTOS DE SALIDA X[k]---\n\n");
	for(i = 0; i<SIGNAL_SIZE;i++)
	{
		printf(" X[%d] = (%f) + (%f)\n",i,hipCrealf(h_Xk[i]),hipCimagf(h_Xk[i]));
	}

	//Se destruye el plan
	hipfftDestroy(plan);

	//Se liberan memorias
	free(h_xn);
	free(h_Xk);
	hipFree(in);
	hipFree(out);
}

